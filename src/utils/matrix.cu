#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iomanip>
#include <iostream>
#include <random>
#include <vector>

#include "time_recorder.h"
#include "util.h"

namespace train_llm {

// naive
__global__ void transpose_kernel1(float* input, float* output, int M, int N) {
  int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = blockIdx.y * blockDim.y + threadIdx.y;

  if (col_idx < N && row_idx < M) {
    int idx = row_idx * N + col_idx;
    int trans_idx = col_idx * M + row_idx;
    output[trans_idx] = input[idx];
  }
}

// float4
__global__ void transpose_kernel_float4(float* input, float* output, int M,
                                        int N) {
  int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = blockIdx.y * blockDim.y + threadIdx.y;

  int float4_N = N >> 2;
  int float4_M = M >> 2;
  if (col_idx < float4_N && row_idx < float4_M) {
    auto float4_off = (row_idx << 2) * N + col_idx << 2;
    // 4 * 1
    const float4* float4_input =
        reinterpret_cast<const float4*>(input + float4_off);

    // 4 * 4
    float4 src_row0 = float4_input[0];
    float4 src_row1 = float4_input[float4_N];
    float4 src_row2 = float4_input[float4_N << 1];
    float4 src_row3 = float4_input[float4_N * 3];

    float4 dst_row0 =
        make_float4(src_row0.x, src_row1.x, src_row2.x, src_row3.x);
    float4 dst_row1 =
        make_float4(src_row0.y, src_row1.y, src_row2.y, src_row3.y);
    float4 dst_row2 =
        make_float4(src_row0.z, src_row1.z, src_row2.z, src_row3.z);
    float4 dst_row3 =
        make_float4(src_row0.w, src_row1.w, src_row2.w, src_row3.w);

    int out_off = (col_idx << 2) * M + row_idx << 2;
    float4* float4_out = reinterpret_cast<float4*>(output + out_off);
    float4_out[0] = dst_row0;
    float4_out[float4_M] = dst_row1;
    float4_out[float4_M << 1] = dst_row2;
    float4_out[float4_M * 3] = dst_row3;
  }
}

// https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
//  对于计算强度比较低的 kernel, 增加线程处理的元素个数即计算强度,
//  一定程度上能增大 GPU 中计算与访存的掩盖, 并配合循环展开提高指令级并行;
// 此外, 由于线程块数量的减少, 减少 GPU 的线程块调度上可能也会带来性能的收益.
template <int BLOCK_SIZE, int NUM_PER_THREAD>
__global__ void __launch_bounds__(1024)
    transpose_kernel_shm(float* input, float* output, int M, int N) {
  const int bx = blockIdx.x, by = blockIdx.y;
  const int tx = threadIdx.x, ty = threadIdx.y;
  __shared__ float shm[BLOCK_SIZE][BLOCK_SIZE + 1];
  int xoff = bx * BLOCK_SIZE + tx;
  int yoff = by * BLOCK_SIZE + ty;
  if (xoff < N) {
#pragma unroll
    for (auto y = 0; y < BLOCK_SIZE; y += blockDim.y) {
      if (yoff + y < M) {
        shm[ty + y][tx] = input[(yoff + y) * N + xoff];
      }
    }
  }
  __syncthreads();
  xoff = by * BLOCK_SIZE + tx;
  yoff = bx * BLOCK_SIZE + ty;
  if (xoff < M) {
#pragma unroll
    for (auto y = 0; y < BLOCK_SIZE; y += blockDim.y) {
      if (yoff + y < N) {
        output[(yoff + y) * M + xoff] = shm[tx][ty + y];
      }
    }
  }
}

// share memory transpose
template <typename T>
__global__ void __launch_bounds__(1024)
    transpose_kernel(const T* src, T* dst, int dstM, int dstN) {
  __shared__ T share_arrary[32][33];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;

  for (int block_offest_y = blockIdx.y * blockDim.y; block_offest_y < dstM;
       block_offest_y += blockDim.y * gridDim.y) {
    for (int block_offest_x = blockIdx.x * blockDim.x; block_offest_x < dstN;
         block_offest_x += blockDim.x * gridDim.x) {
      // src coordinate
      int src_col = block_offest_y + tx;
      int src_row = block_offest_x + ty;

      if (src_col < dstM && src_row < dstN) {
        share_arrary[ty][tx] = src[src_row * dstM + src_col];  // 合并访存
      }
      __syncthreads();
      // dst coordinate
      // Block thread的坐标映射是根据 dst来着
      int dst_row = block_offest_y + ty;
      int dst_col = block_offest_x + tx;
      if (dst_row < dstM && dst_col < dstN) {
        dst[dst_row * dstN + dst_col] = share_arrary[tx][ty];
      }
    }
  }
}

void transpose(float* input, float* output, int M, int N) {
  {
    TimeRecorder("transpose1");
    dim3 block(4, 64);
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);
    transpose_kernel1<<<grid, block>>>(input, output, M, N);
  }
  {
    TimeRecorder("transpose2");
    dim3 block(16, 16);
    dim3 grid((N >> 2 + block.x - 1) / block.x,
              (M >> 2 + block.y - 1) / block.y);
    transpose_kernel_float4<<<grid, block>>>(input, output, M, N);
  }
  {
    TimeRecorder("transpose_shm");
    const int NUM_PER_THREAD = 4;
    dim3 block(32, 32 / NUM_PER_THREAD);
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);
    transpose_kernel_shm<32, NUM_PER_THREAD>
        <<<grid, block>>>(input, output, M, N);
  }
  {
    TimeRecorder("transpose_shm2");
    const dim3 block(32, 32);
    const dim3 grid((N + 31) / 32, (M + 31) / 32);
    transpose_kernel<<<grid, block>>>(input, output, N, M);
  }
}
}  // namespace train_llm

using namespace train_llm;
// test
int main() {
  int M = 8196;
  int N = 8196;
  size_t len = M * N;
  std::vector<float> data(len, 0);
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(0, 1.0);
  for (auto i = 0; i < M; ++i) {
    for (auto j = 0; j < N; ++j) {
      data[i * N + j] = dis(gen);
    }
  }
  float* d_mat = nullptr;
  float* d_output = nullptr;
  hipMalloc((void**)&d_mat, len * sizeof(float));
  hipMalloc((void**)&d_output, len * sizeof(float));
  hipMemcpy(d_mat, data.data(), len * sizeof(float), hipMemcpyHostToDevice);
  train_llm::transpose(d_mat, d_output, M, N);
  // debug_display_mat("origin mat", d_mat, M, N);
  // debug_display_mat("transpose mat", d_output, N, M);
  return 0;
}